#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : mce.cu
 Author      : Jesse Harder
 Supervisor	 : Dr. Christopher Henry, P. Eng.
 Date	     : August 23, 2016
 Version	 : 1.0
 Description : This program will find the maximal clique enumerations
 	 	 	 	 for a given Adjacency Matrix (Calculated on a GPU device)
 License	 : Licensed under the Non-Profit Open Software License version 3.0
 1) Grant of Copyright License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, for the duration of the copyright, to do the following:

 a) to reproduce the Original Work in copies, either alone or as part of a collective work;

 b) to translate, adapt, alter, transform, modify, or arrange the Original Work, thereby
 creating derivative works ("Derivative Works") based upon the Original Work;

 c) to distribute or communicate copies of the Original Work and Derivative Works
 to the public, with the proviso that copies of Original Work or Derivative Works
 that You distribute or communicate shall be licensed under this Non-Profit Open Software
 License or as provided in section 17(d);

 d) to perform the Original Work publicly; and

 e) to display the Original Work publicly.

 2) Grant of Patent License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, under patent claims owned or controlled by
  the Licensor that are embodied in the Original Work as furnished by the Licensor,
  for the duration of the patents, to make, use, sell, offer for sale, have made,
   and import the Original Work and Derivative Works.

 3) Grant of Source Code License. The term "Source Code" means the preferred
 form of the Original Work for making modifications to it and all available
 documentation describing how to modify the Original Work. Licensor agrees to
 provide a machine-readable copy of the Source Code of the Original Work along
 with each copy of the Original Work that Licensor distributes. Licensor reserves
 the right to satisfy this obligation by placing a machine-readable copy of the
 Source Code in an information repository reasonably calculated to permit
 inexpensive and convenient access by You for as long as Licensor continues
 to distribute the Original Work.

 4) Exclusions From License Grant. Neither the names of Licensor, nor the names
 of any contributors to the Original Work, nor any of their trademarks or service
 marks, may be used to endorse or promote products derived from this Original Work
 without express prior permission of the Licensor. Except as expressly stated
  herein, nothing in this License grants any license to Licensor's trademarks,
  copyrights, patents, trade secrets or any other intellectual property. No patent
  license is granted to make, use, sell, offer for sale, have made, or import embodiments
  of any patent claims other than the licensed claims defined in Section 2. No license
 is granted to the trademarks of Licensor even if such marks are included in the Original
  Work. Nothing in this License shall be interpreted to prohibit Licensor from licensing
  under terms different from this License any Original Work that Licensor otherwise would
  have a right to license.

 5) External Deployment. The term "External Deployment" means the use, distribution, or
 communication of the Original Work or Derivative Works in any way such that the Original
 Work or Derivative Works may be used by anyone other than You, whether those works are
 distributed or communicated to those persons or made available as an application intended
 for use over a network. As an express condition for the grants of license hereunder,
 You must treat any External Deployment by You of the Original Work or a Derivative
 Work as a distribution under section 1(c).

 6) Attribution Rights. You must retain, in the Source Code of any Derivative Works
 that You create, all copyright, patent, or trademark notices from the Source Code of
 the Original Work, as well as any notices of licensing and any descriptive text
 identified therein as an "Attribution Notice." You must cause the Source Code for
 any Derivative Works that You create to carry a prominent Attribution Notice reasonably
 calculated to inform recipients that You have modified the Original Work.

 7) Warranty of Provenance and Disclaimer of Warranty. The Original Work is provided
 under this License on an "AS IS" BASIS and WITHOUT WARRANTY, either express or implied,
 including, without limitation, the warranties of non-infringement, merchantability or
 fitness for a particular purpose. THE ENTIRE RISK AS TO THE QUALITY OF THE ORIGINAL WORK
 IS WITH YOU. This DISCLAIMER OF WARRANTY constitutes an essential part of this License.
 No license to the Original Work is granted by this License except under this disclaimer.

 8) Limitation of Liability. Under no circumstances and under no legal theory, whether
 in tort (including negligence), contract, or otherwise, shall the Licensor be liable
 to anyone for any direct, indirect, special, incidental, or consequential damages of
 any character arising as a result of this License or the use of the Original Work
 including, without limitation, damages for loss of goodwill, work stoppage, computer
 failure or malfunction, or any and all other commercial damages or losses. This limitation
 of liability shall not apply to the extent applicable law prohibits such limitation.

 9) Acceptance and Termination. If, at any time, You expressly assented to this License,
 that assent indicates your clear and irrevocable acceptance of this License and all of
 its terms and conditions. If You distribute or communicate copies of the Original Work
 or a Derivative Work, You must make a reasonable effort under the circumstances to obtain
 the express assent of recipients to the terms of this License. This License conditions
 your rights to undertake the activities listed in Section 1, including your right to create
 Derivative Works based upon the Original Work, and doing so without honoring these terms and
 conditions is prohibited by copyright law and international treaty. Nothing in this License
 is intended to affect copyright exceptions and limitations (including "fair use" or "fair
 dealing"). This License shall terminate immediately and You may no longer exercise any of
 the rights granted to You by this License upon your failure to honor the conditions in Section 1(c).

 10) Termination for Patent Action. This License shall terminate automatically and You
 may no longer exercise any of the rights granted to You by this License as of the date
 You commence an action, including a cross-claim or counterclaim, against Licensor or any
 licensee alleging that the Original Work infringes a patent. This termination provision
 shall not apply for an action alleging patent infringement by combinations of the Original
  Work with other software or hardware.

 11) Jurisdiction, Venue and Governing Law. Any action or suit relating to this License
 may be brought only in the courts of a jurisdiction wherein the Licensor resides or in
 which Licensor conducts its primary business, and under the laws of that jurisdiction
 excluding its conflict-of-law provisions. The application of the United Nations Convention
 on Contracts for the International Sale of Goods is expressly excluded. Any use of the Original
 Work outside the scope of this License or after its termination shall be subject to the
 requirements and penalties of copyright or patent law in the appropriate jurisdiction.
 This section shall survive the termination of this License.

 12) Attorneys' Fees. In any action to enforce the terms of this License or seeking
 damages relating thereto, the prevailing party shall be entitled to recover its costs and
 expenses, including, without limitation, reasonable attorneys' fees and costs incurred in
 connection with such action, including any appeal of such action. This section shall survive
 the termination of this License.

 13) Miscellaneous. If any provision of this License is held to be unenforceable, such provision
 shall be reformed only to the extent necessary to make it enforceable.

 14) Definition of "You" in This License. "You" throughout this License, whether in upper or
 lower case, means an individual or a legal entity exercising rights under, and complying with
 all of the terms of, this License. For legal entities, "You" includes any entity that controls,
 is controlled by, or is under common control with you. For purposes of this definition, "control"
 means (i) the power, direct or indirect, to cause the direction or management of such entity,
 whether by contract or otherwise, or (ii) ownership of fifty percent (50%) or more of the outstanding
 shares, or (iii) beneficial ownership of such entity.

 15) Right to Use. You may use the Original Work in all ways not otherwise restricted or conditioned
  by this License or by law, and Licensor promises not to interfere with or be responsible for such uses by You.

 16) Modification of This License. This License is Copyright © 2005 Lawrence Rosen.
 Permission is granted to copy, distribute, or communicate this License without modification.
 Nothing in this License permits You to modify this License as applied to the Original Work or to
 Derivative Works. However, You may modify the text of this License and copy, distribute or communicate
 your modified version (the "Modified License") and apply it to other original works of authorship
 subject to the following conditions: (i) You may not indicate in any way that your Modified License
 is the "Open Software License" or "OSL" and you may not use those names in the name of your Modified
 License; (ii) You must replace the notice specified in the first paragraph above with the notice
 "Licensed under <insert your license name here>" or with a notice of your own that is not confusingly
 similar to the notice in this License; and (iii) You may not claim that your original works are open
 source software unless your Modified License has been approved by Open Source Initiative (OSI) and
 You comply with its license review and certification process.

 17) Non-Profit Amendment. The name of this amended version of the Open Software License ("OSL 3.0")
 is "Non-Profit Open Software License 3.0". The original OSL 3.0 license has been amended as follows:

 (a) Licensor represents and declares that it is a not-for-profit organization that derives no revenue
 whatsoever from the distribution of the Original Work or Derivative Works thereof, or from support
 or services relating thereto.

 (b) The first sentence of Section 7 ["Warranty of Provenance"] of OSL 3.0 has been stricken. For
 Original Works licensed under this Non-Profit OSL 3.0, LICENSOR OFFERS NO WARRANTIES WHATSOEVER.

 (c) In the first sentence of Section 8 ["Limitation of Liability"] of this Non-Profit OSL 3.0,
 the list of damages for which LIABILITY IS LIMITED now includes "direct" damages.

 (d) The proviso in Section 1(c) of this License now refers to this "Non-Profit Open Software
 License" rather than the "Open Software License". You may distribute or communicate the Original
 Work or Derivative Works thereof under this Non-Profit OSL 3.0 license only if You make the
 representation and declaration in paragraph (a) of this Section 17. Otherwise, You shall distribute or
 communicate the Original Work or Derivative Works thereof only under the OSL 3.0 license and You shall
 publish clear licensing notices so stating. Also by way of clarification, this License does not authorize
 You to distribute or communicate works under this Non-Profit OSL 3.0 if You received them under
 the original OSL 3.0 license.

 (e) Original Works licensed under this license shall reference "Non-Profit OSL 3.0"
 in licensing notices to distinguish them from works licensed under the original OSL 3.0 license.
 ============================================================================
 */

#include <iostream>
#include <stdlib.h>
#include <fstream>		//Read input and write output files
#include <string>		//Provides string object
#include <sstream>		//Provides methods for working with strings
#include <thrust/scan.h>//Provides parallel prefix scan algorithm

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

texture<int, 1, hipReadModeElementType> data_texture;
//Number of input vectors, specified by user
unsigned int VECTORS = 0;
//Highest amount of neighbours any one vertex has, as calculated by the program
unsigned int maxNeighbours = 0;

//This class is to be inherited and will automatically cudaMallocManage (allocate in unified memory)
// when a new child class is instantiated
class ManagedStruct
{
	public:
		void *operator new(size_t len){
			//Instantiate object and return pointer to unified memory address
			void *ptr;
			hipMallocManaged(&ptr, len);
			return ptr;
		}

		void operator delete(void *ptr){
			hipFree(ptr);
		}
};

//This structure represents a graph node, containing the 3 arrays a graph node consists of
// When instantiated, it will automatically allocate itself to shared memory
struct graphNode : public ManagedStruct{
	//A list of ids of candidate nodes to match with
	int* cand;
	//A list of ids of nodes that would be redundant to match with
	int* cnot;
	//A list of ids in the current clique being generated
	int* compsub;
};

/******************************************************************************
 * isConnectedGPU (GPU function)
 *
 * This function will determine if one int is found in a list of other
 * ints. Each int represents the id of a vertex
 *
 * [in]:
 * 		indexA: The first vertex id, the value to be looked for
 * 		indexB: The second vertex id, the index of the list to be searched though
 * 		maxNeighbours: The length of a row in data
 *
 * [out]:
 *		None
 *
 * [return]:
 * 		True if the value is found, else false
 *
 * [notes]:
 * 		Note that this function makes use of the adjacency array which has been
 * 		bound to texture memory on the device
*******************************************************************************/
__device__ bool isConnectedGPU(int indexA, int indexB, unsigned maxNeighbours){

	// Check if the number, indexA, exists in the indexB'th row of the array, data
	unsigned index = (maxNeighbours + 1) * indexB;

	for(unsigned i = 1; i <= maxNeighbours; ++i){
		//Check if the given element in the texture array is the element we're looking for
		if(tex1Dfetch(data_texture, (index +  i)) == indexA)
			//Element was found
			return true;
	}

	//Element not found
	return false;
}

/******************************************************************************
 * createNodes (GPU kernel function)
 *
 * This function will create the next level of nodes for the MCE algorithm,
 * or output the node's compsub, if it won't produce child nodes.
 * Each thread in this algorithm represents one node
 * This function closely follows the Recursive Clique Enumeration algorithm
 * provided in
 * "A scalable, parallel algorithm for maximal clique enumeration" by
 * Matthew C. Schmidt, Nagiza F. Samatova, Kevin Thomas, and Byung-Hoon Park
 *
 * [in]:
 * 		inNode: A pointer to an array of input nodes
 * 		outNode: A pointer to an array of nodes used for output
 * 		sizeArray: A prefix summed array that tells each thread where in
 * 			outNode to write it's results
 * 		maxNeighbours: The highest number of neighbours a node can possible have
 * 		workSize: The number of nodes to be operated on
 *
 * [out]:
 *		outNode: Each node produced will be written to this array
 *
 * [return]:
 * 		Void
*******************************************************************************/
__global__ void createNodes(graphNode** inNode, graphNode** outNode, unsigned* sizeArray, unsigned maxNeighbours, unsigned workSize){

	//The id of this thread
	unsigned id = threadIdx.x + (blockDim.x * blockIdx.x);
	//fixp: The vertex in cand connected to the highest number of other vertices in cand
	int fixp;
	//cur_v: The vertex currently being worked with (initially fixp)
	int cur_v;
	//Get the location where this thread will write its output
	unsigned outputIndex;


	if(id < workSize){
		outputIndex = sizeArray[id];
		int record = -1;
		//If cand and cnot are empty, write this inNode's compsub list to the output array
		if(inNode[id]->cand[0] > -1){

			//Get the vertex in cand connected to the most other vertices in cand
			for(unsigned i = 0; i < maxNeighbours; ++i){
				int count = 0;
				for(unsigned j = 0; j < maxNeighbours; ++j){
					//Check if the current vertex i is connected to vertix j
					if(inNode[id]->cand[i] != -1 && inNode[id]->cand[j] != -1 && isConnectedGPU(inNode[id]->cand[i], inNode[id]->cand[j], maxNeighbours)){
						count++;
					}
				}
				//If the current vertex has the  most connections, then it becomes fixp
				if(count > record){
					record = count;
					fixp = inNode[id]->cand[i];
				}
			}
			//Set current vector to the vector connected to the most other vectors in cand. vector
			cur_v = fixp;

			//Track how many nodes this thread has generated so far, so we know where to write out output
			unsigned outputId = 0;

			while(cur_v != -1){

				unsigned count = 0;

				//get newnot
				//newnot = all elements in not connected to fixp
				for(unsigned i = 0; i < maxNeighbours; ++i){
					if(inNode[id]->cnot[i] > -1 && isConnectedGPU(inNode[id]->cnot[i], cur_v, maxNeighbours)){
						outNode[outputIndex + outputId]->cnot[count] = inNode[id]->cnot[i];
						count++;
					}
				}

				count = 0;
				//Get newcand
				// newCand = all nodes in cand connected to cur_v
				for(unsigned i = 0; i < maxNeighbours; ++i){
					if(inNode[id]->cand[i] > -1 && isConnectedGPU(inNode[id]->cand[i], cur_v, maxNeighbours)){
						outNode[outputIndex + outputId]->cand[count] = inNode[id]->cand[i];
						count++;
					}
				}

				//get newcompsub
				//newcompsub = oldcompsub + cur_v
				for(unsigned i = 0; i <maxNeighbours; ++i){
					if(inNode[id]->compsub[i] != -1){
						outNode[outputIndex + outputId]->compsub[i] = inNode[id]->compsub[i];
					}else{
						outNode[outputIndex + outputId]->compsub[i] = cur_v;
						break;
					}
				}

				//add cur_v to cnot
				for(unsigned i = 0; i <maxNeighbours; ++i){
					if(inNode[id]->cnot[i] == -1){
						inNode[id]->cnot[i] = cur_v;
							break;
					}
				}


				//remove cur_v from cand
				for(unsigned i = 0; i <maxNeighbours; ++i){
					if(inNode[id]->cand[i] == cur_v){
						inNode[id]->cand[i] = -1;
						break;
					}
				}

				cur_v = -1;
				//Attempt to find next cur_v (the while loop terminates if cur_v remains -1)
				//Next cur_v is the first element in cand that is not connected to fixp
				for(unsigned i = 0; i < maxNeighbours; ++i){
					if(inNode[id]->cand[i] > -1 && !isConnectedGPU(inNode[id]->cand[i], fixp, maxNeighbours)){
						outputId++;
						cur_v = inNode[id]->cand[i];
						break;
					}
				}
			}
		}
	}
	__syncthreads();

}

/******************************************************************************
 * sizeKernel (GPU kernel function)
 *
 * This function will determine how many child nodes each input node will produce.
 * Each thread handles one input node
 *
 * [in]:
 * 		nodes: A pointer to the array of input nodes
 * 		sizeArray: A pointer to the output array for the calculated sizes
 * 		maxNeighbours: The highest number of neighbours a node may have
 * 		workSize: The number of nodes in nodes to be operated on
 *
 * [out]:
 *		Each thread will write the number of children its corresponding node
 *			will produce to the thread id'th spot in sizeArray
 *
 * [return]:
 * 		Void
*******************************************************************************/
__global__ void sizeKernel(graphNode** nodes, unsigned* sizeArray, unsigned maxNeighbours, unsigned workSize){

	//Shared int array used for tracking the input node's cand vector
	extern __shared__ int shCand[];

	unsigned id = threadIdx.x + (blockDim.x * blockIdx.x);
	unsigned index = threadIdx.x  * maxNeighbours;

	//Final result, how many nodes the input node will produce
	unsigned size = 0;

	//the vertex connected to input node, connected to the most other vertices that are connected to input node
	int fixp;
	//Current node being handled
	int cur_v;
	//Store the highest amount of connections any vertex in cand has
	int record = -1;

	if(id < workSize){
		//Load cand into shared memory
		for(unsigned i = 0; i < maxNeighbours; ++i){
			shCand[index + i] = nodes[id]->cand[i];
		}

		if(shCand[index] > -1){

			//Find fixp
			//For every vertex in cand, check if that vertex is connected to any of the other vertices in cand
			for(unsigned i = 0; i < maxNeighbours; ++i){
				//Track how many matches this vertex has made
				int count = 0;

				//Check if the current vertex is in any of the other vertices in cand
				for(unsigned j = 0; j < maxNeighbours; ++j){
					if(shCand[index + i] != -1 && shCand[index + j] != -1 && isConnectedGPU(shCand[index + i], shCand[index + j], maxNeighbours)){
						count++;
					}
				}

				//If the current node has the highest count, record the count and set fixp to that node id
				if(count > record){
					record = count;
					fixp = shCand[index + i];
				}
			}


			cur_v = fixp;
			//Every iteration of this loop means one more node must be created
			while(cur_v != -1){
				size++;

				//Remove cur_v from cand
				for(unsigned i = 0; i < maxNeighbours; ++i){
					if(shCand[index + i] == cur_v)
						shCand[index + i] = -1;
				}

				cur_v = -1;
				//Find the next vertex in cand that is not connected to fixp, exit (cur_v = -1) if none are found
				for(unsigned i = 0; i < maxNeighbours; ++i){
					if(shCand[index + i] > -1 && !isConnectedGPU(shCand[index + i], fixp, maxNeighbours)){
						cur_v = shCand[index + i];
						break;
					}
				}
			}
		}
	}


	//Sync threads then write results to the output array
	__syncthreads();
	if(id < workSize){
		sizeArray[id] = size;
	}
}

/******************************************************************************
 * isConnected
 *
 * This function will determine if an integer value exists in a row of a given
 * array.
 *
 * [in]:
 * 		value: The value to be searched for
 * 		row: The row to be searched though
 * 		data: The array containing the data to be searched
 *
 * [out]:
 *		None
 *
 * [return]:
 * 		True if the value is found, else false
*******************************************************************************/
bool isConnected(unsigned value, int* data, int row){
	for(unsigned i = 1; i <= maxNeighbours; i++){
		if(data[row * (maxNeighbours + 1) + i] == value){
			return true;
		}
	}
	return false;
}

/******************************************************************************
 * writeToFile
 *
 * This function will write the compsub array values of all nodes
 * that have empty cand and cnot arrays to cliques.txt
 *
 * [in]:
 * 		data: The array of graphNodes to potentially be written out
 * 		j: The current maximum size of a compsub array
 * 		out: An output stream for writing to the text file
 * 		workSize: Number of nodes in "data"
 *
 * [out]:
 *		The compsubs of all finished nodes written to cliques.txt
 *
 * [return]:
 * 		Void
 *
 * [notes]:
 * 		If the first element of cand, cnot, or compsub are -1,
 * 			all elements are -1 in that respective array
 * 		-1's will not be written to the text file
*******************************************************************************/
void writeToFile(graphNode** data, unsigned j, std::ofstream& out, unsigned workSize){

	for(unsigned i = 0; i < workSize; ++i){
		if(data[i]->cand[0] == -1 && data[i]->cnot[0] == -1){
			for(unsigned k = 0; k < j; ++k){
				out << data[i]->compsub[k] << " ";
			}
			out << std::endl;
		}
	}
}

/******************************************************************************
 * maximalCliqueEnumeration
 *
 * This function will perform the algorithm to generate all maximal cliques.
 *
 * [in]:
 * 		structArray: Array of structs holding the first level of nodes, will
 * 			also be used to write future nodes to
 * 		initNodeCount: The initial size of struct array
 * 		intData: An array containing the initial input adjacency values
 * 		maxTileWidth: A calculated value based on shared memory limits to determine
 * 			how large out sizeKernels can be
 *
 * [out]:
 *		Calls writeToFile to write out the cliques
 *
 * [return]:
 * 		Void
*******************************************************************************/
void maximalCliqueEnumeration(graphNode** structArray, unsigned initNodeCount, int* intData, unsigned maxTileWidth, bool singleCliques){

	//Declare array for holding the counts of the amounts of nodes to be generated
	unsigned prefixArraySize = VECTORS * 3;
	unsigned* prefixSummed = new unsigned[prefixArraySize];

	//Allocate and copy node size count array to device (known as prefixSummed on host)
	unsigned* dX;
	CUDA_CHECK_RETURN(hipMalloc((void**)&dX, prefixArraySize * sizeof(unsigned)));
	CUDA_CHECK_RETURN(hipMemcpy(dX, prefixSummed, prefixArraySize * sizeof(unsigned), hipMemcpyHostToDevice));

	//Allocate and copy Adjacency data to device (known as intData on host)
	int* dData;
	CUDA_CHECK_RETURN(hipMalloc((void**)&dData, VECTORS * (maxNeighbours + 1) * sizeof(int)));
	CUDA_CHECK_RETURN(hipMemcpy(dData, intData, VECTORS * (maxNeighbours + 1) * sizeof(int), hipMemcpyHostToDevice));

	//As intData/dData is only ever read from the GPU (Never written to on GPU, and never read from/ written to past this point
	//	on the CPU) we bind it to texture memory on the device
	hipBindTexture(0, data_texture, dData, VECTORS * (maxNeighbours + 1) * sizeof(int));
	printf("%s\n", hipGetErrorString(hipGetLastError()));

	std::cout << "Performing maximal clique enumeration ..." << std::endl;

	//Cliques are formed by a tree, generated one level at a time
	graphNode** structArray2;
	unsigned workSize = initNodeCount;

	bool swap = false;	//Determine which half of loop is run (Switches on every iteration)

	// Track number of passes through the loop
	unsigned j = 0;

	unsigned i = 0;

	//Output file stream
	std::ofstream out("cliques.txt");
	do{
		//Get launch configurations for createNodes
		unsigned dimBlock = min(workSize + 1 + (32 - ((workSize + 1) % 32)), 512);
		unsigned dimGrid = ceil((float)workSize / dimBlock);
		//Get launch configuration for sizeKernel
		unsigned sizeKernelBlock = min(maxTileWidth, dimBlock);
		unsigned sizeKernelGrid = ceil((float)workSize / sizeKernelBlock);
		unsigned sharedMemSize = maxNeighbours * sizeKernelBlock * sizeof(int);
		//std::cout << j << "th loop!\n";

		if(swap == false){
			//Launch a kernel that will determine how many nodes each of the current nodes will generate
			sizeKernel <<< sizeKernelGrid, sizeKernelBlock, sharedMemSize >>>(structArray, dX, maxNeighbours, workSize);
			//printf("%s\n", hipGetErrorString(hipGetLastError()));
			CUDA_CHECK_RETURN(hipDeviceSynchronize());

			//Output the compsubs of completed nodes to a text file
			//If user does not want cliques of single nodes, skip this on the first pass through
			if(j > 0 || singleCliques == true){
				writeToFile(structArray, j + 1, out, workSize);
			}

			//Copy the node sizes back from the device
			CUDA_CHECK_RETURN(hipMemcpy(prefixSummed, dX, (workSize + 1) * sizeof(unsigned), hipMemcpyDeviceToHost));

			//Call thrust library to prefix sum the node counts
			thrust::exclusive_scan(prefixSummed, prefixSummed + prefixArraySize, prefixSummed);

			//Put prefix summed node sizes back on the device
			CUDA_CHECK_RETURN(hipMemcpy(dX, prefixSummed, (workSize + 1) * sizeof(unsigned), hipMemcpyHostToDevice));


			//Allocate the memory for the nodes to be generated
			//There should be a number of nodes equal to the sum of all the results from sizeKernel
			//If no nodes are to be generated, we are done; exit
			//std::cout << "These " << workSize << " nodes will generate " << prefixSummed[workSize] <<" more nodes!\n";
			if(prefixSummed[workSize] == 0){
				hipFree(structArray);
				break;
			}

			structArray2 = new graphNode*[prefixSummed[workSize]];
			CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray2), sizeof(structArray2) * (prefixSummed[workSize])));

			for(i = 0; i < prefixSummed[workSize]; ++i){
				//Allocate node and its arrays, and initialize the arrays to -1
				structArray2[i] = new graphNode;
				structArray2[i]->cand = new int[maxNeighbours];
				CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray2[i]->cand), sizeof(int) * maxNeighbours));
				structArray2[i]->cnot = new int[maxNeighbours];
				CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray2[i]->cnot), sizeof(int) * maxNeighbours));
				structArray2[i]->compsub = new int[maxNeighbours];
				CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray2[i]->compsub), sizeof(int) * maxNeighbours));

				for(unsigned k = 0; k < maxNeighbours; ++k){
					structArray2[i]->cand[k] = -1;
					structArray2[i]->cnot[k] = -1;
					structArray2[i]->compsub[k] = -1;
				}
			}
			CUDA_CHECK_RETURN(hipDeviceSynchronize());

			//Call kernel to create next level of nodes
			createNodes <<< dimGrid, dimBlock >>>(structArray, structArray2, dX, maxNeighbours, workSize);

			//Free structArray 1, so it can be reallocated for output on next pass
			hipFree(structArray);
			//Run other half of algorithm on next pass
			swap = true;

			CUDA_CHECK_RETURN(hipDeviceSynchronize());

		}else{
			//This segment is the same as the other half of the if statement, but structArray and structArray2 are switched

			//Launch a kernel that will determine how many nodes each of the current nodes will generate
			sizeKernel <<< sizeKernelGrid, sizeKernelBlock, sharedMemSize >>>(structArray2, dX, maxNeighbours, workSize);
			CUDA_CHECK_RETURN(hipDeviceSynchronize());

			//Write completed nodes' compsubs to the output file
			writeToFile(structArray2, j + 1, out, workSize);

			//Copy the node sizes back from the device
			CUDA_CHECK_RETURN(hipMemcpy(prefixSummed, dX, (workSize +1) * sizeof(unsigned), hipMemcpyDeviceToHost));

			//Call thrust library to prefix sum the node counts
			thrust::exclusive_scan(prefixSummed, prefixSummed + prefixArraySize, prefixSummed);

			//Put prefix summed node sizes back on the device
			CUDA_CHECK_RETURN(hipMemcpy(dX, prefixSummed, (workSize + 1) * sizeof(unsigned), hipMemcpyHostToDevice));

			//Allocate the memory for the nodes to be generated
			//There should be a number of nodes equal to the sum of all the results from sizeKernel
			//If no nodes are to be generated, we are done; exit
			//std::cout << "These " << workSize << " nodes will generate " << prefixSummed[workSize] <<" more nodes!\n";
			if(prefixSummed[workSize] == 0){
				hipFree(structArray2);
				break;
			}
			structArray = new graphNode*[prefixSummed[workSize]];
			CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray), sizeof(structArray) * (prefixSummed[workSize])));
			for(i = 0; i < prefixSummed[workSize]; ++i){

				structArray[i] = new graphNode;
				structArray[i]->cand = new int[maxNeighbours];
				CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray[i]->cand), sizeof(int) * maxNeighbours));
				structArray[i]->cnot = new int[maxNeighbours];
				CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray[i]->cnot), sizeof(int) * maxNeighbours));
				structArray[i]->compsub = new int[maxNeighbours];
				CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray[i]->compsub), sizeof(int) * maxNeighbours));

				for(unsigned k = 0; k < maxNeighbours; ++k){
					structArray[i]->cand[k] = -1;
					structArray[i]->cnot[k] = -1;
					structArray[i]->compsub[k] = -1;
				}
			}
			CUDA_CHECK_RETURN(hipDeviceSynchronize());

			//Call kernel to create next level of nodes
			createNodes <<< dimGrid, dimBlock >>>(structArray2, structArray, dX, maxNeighbours, workSize);

			//Free structArray 2 so it can be used for output on next pass
			hipFree(structArray2);

			//Run other half of algorithm on next pass
			swap = false;

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
		}
		//Set the number of nodes to be operated upon on the next pass
		workSize = prefixSummed[workSize];
		j++;

	//Loop again, unless no new nodes are to be generated (Last element of prefix sum is zero)
	}while(true);

}


/******************************************************************************
 * generateInitialNodes
 *
 * This function will generate the first level of nodes for the maximal clique
 * enumeration. It will also determine the number of nodes generated.
 *
 *
 * [in]:
 * 		structArray: A pointer to an array of graphNode structures, where the
 * 			generated nodes will be written to
 * 		intData: A pointer to the array of adjacency values
 * 		fixp: The id of the vertex/node that neighbours the most of other nodes
 *
 * [out]:
 *		structArray will have all generated nodes written to it
 *
 * [return]:
 * 		initNodeCount: The count of nodes generated
 *
 * [notes]:
 * 		This function follows the recursive clique enumeration algorithm from
 * 			"A scalable, parallel algorithm for maximal clique enumeration"
 *
*******************************************************************************/
unsigned generateInitialNodes(graphNode** structArray, int* intData, unsigned fixp){
	//cur_v is the current vertex being worked with, initially fixp
	int cur_v = fixp;
	//std::cout << "fixp: "<< fixp << std::endl;

	std::cout << "Generating initial nodes ..." << std::endl;


	//Two temporary arrays used to create the initial nodes
	int* initCand = new int[VECTORS];
	int* initNot = new int[VECTORS];
	for(unsigned i = 0; i < VECTORS; ++i){
		//Initial cand is all values 0 to VECTORS-1
		initCand[i] = i;
		//Initial not is empty (all -1)
		initNot[i] = -1;
	}


	//Pointer to array of structures used to represent graph nodes
	unsigned initNodeCount = 0;
	while(cur_v >= 0){
		//Declare new graphNode and allocate its arrays into unified memory
		structArray[initNodeCount] = new graphNode;

		structArray[initNodeCount]->cand = new int[maxNeighbours];
		CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray[initNodeCount]->cand), sizeof(int) * maxNeighbours));

		structArray[initNodeCount]->cnot = new int[maxNeighbours];
		CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray[initNodeCount]->cnot), sizeof(int) * maxNeighbours));

		structArray[initNodeCount]->compsub = new int[maxNeighbours];
		CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray[initNodeCount]->compsub), sizeof(int) * maxNeighbours));

		for(unsigned k = 0; k < maxNeighbours; ++k){
			//Initialize all array values to null (-1)
			structArray[initNodeCount]->cand[k] = -1;
			structArray[initNodeCount]->cnot[k] = -1;
			structArray[initNodeCount]->compsub[k] = -1;
		}

		//Fill in the 'not' values
		unsigned notCount = 0;
		for(unsigned i = 0; i < initNodeCount; ++i){
			if(initNot[i] > -1 && isConnected(initNot[i], intData, cur_v)){
				structArray[initNodeCount]->cnot[notCount] = initNot[i];
				notCount++;
			}
		}

		//Fill in the 'cand' values
		unsigned candCount = 0;
		for(unsigned i = 0; i < VECTORS; ++i){
			if(initCand[i] > -1 && isConnected(initCand[i], intData, cur_v)){
				structArray[initNodeCount]->cand[candCount] = initCand[i];
				candCount++;
			}
		}
		//Initial compsub is just cur_v
		structArray[initNodeCount]->compsub[0] = cur_v;

		//Remove cur_v from cand, add it to not
		initNot[initNodeCount] = cur_v;
		initCand[cur_v] = -1;

		//Find the next vertex to operate upon
		//This vertex must be in cand and not connected to fixp
		for(unsigned i = 0; i < VECTORS; ++i){
			if(initCand[i] > -1 && !isConnected(initCand[i], intData, fixp)){
				cur_v = i;
				break;
			}else{
				cur_v = -1;
			}
		}
		initNodeCount++;
	}

	return initNodeCount;
}

/******************************************************************************
 * genereateAdjacencyValues
 *
 * This function will populate an array with its adjacency values
 * Each row represents one vertex
 * The first element in a row is the count of how many neighbours that vertex has
 * Each following element is an int id of a neighbouring vertex
 * -1's represent nulls, or no neighbours
 * The neighbours are determines by reading in a bit matrix file, wherein
 * each 1 in a row represents a neighbour
 *
 * [in]:
 * 		intData: A pointer to the array to write output to
 * 		inFile: A string containing the name of the file to read
 *
 * [out]:
 *		intData with all adjacency values written to it
 *
 * [return]:
 * 		Void
 *
 * [notes]:
 * 		maxNeighbours is previously determined to be the most amount of
 * 		neighbours any one node can have, thus maxNeighbours + 1 is the max
 * 		size of a row in intData
 *
*******************************************************************************/
void generateAdjacencyValues(int* intData, std::string inFile){
	std::fstream inputFile(inFile.c_str(), std::ios_base::in);
	if(inputFile.fail()){
		std::cerr << "Error: Adjacency Matrix file, " << inFile.c_str() << ", could not be found." << std::endl;
		exit(1);
	}
	unsigned char c;
	unsigned i = 0;
	unsigned counter = 0;
	//Fill in the values of intData, using the input adjacency matrix file
	//Counter tracks how many neighbours this vertex has
	while(inputFile >> c){
		if(i % VECTORS == 0 && i > 0){
			intData[(maxNeighbours + 1) * ((i / VECTORS) - 1)] = counter;
			counter = 0;
		}

		// c - 48 gets numeric value of '0' and '1' chars
		if(c - 48 == 1){
			intData[(i / VECTORS) * (maxNeighbours + 1) + 1 + counter] = i % VECTORS;
			counter++;
		}
		i++;
	}

}

/******************************************************************************
 * printHelp
 *
 * Prints out all available command parameters, and a short description of each
 *
 * [in]:
 * 		None
 * [out]:
 * 		A list and description of all command parameters
 *
 * [return]:
 * 		void
 *
 *******************************************************************************/
void printHelp(){
	printf("Required Parameters:\n");
	printf("\t-v [int > 0]: Specify how many vertices the adjacency matrix has. !!Required Parameter!!\n");

	printf("Optional Parameters:\n");
	printf("\t-f [complete path and file name]: Specifies which file houses the input adjacency matrix.\n");
	printf("\t-gpu [int >= 0]: Specify which device to run GPU segments on. Requires a valid device id.\n");
	printf("\t-help: Prints out available command line parameters, then exits program\n");
}

/******************************************************************************
 * mce.cu
 *
 * This program will find all of the maximal cliques given an adjacency matrix.
 * This is an GPU based implementation of the Recursive Clique Enumerate function found in
 *  "A scalable, parallel algorithm for maximal clique enumeration" by
 *  Matthew C. Schmidt, Nagiza F. Samatova, Kevin Thomas, and Byung-Hoon Park
 *  (I recommend getting an understanding of the Recursice Clique Enumeration
 *  	algorithm presented in this article before trying to read this code)
 *
 *
 *
 * [Command line parameters]:
 * 		-v [int > 0]: The number of vertices in the input file
 * 			!!!! This parameter is mandatory at run time !!!!
 *		-gpu [int >= 0]: Specify which GPU device to run device code on
 *			(must be a valid device id)
 *		-f [full path and file name]:
 *			Allows the user to specify a file other than adjMatrix.txt
 *			to be used for input
 *		-help: Prints out all command parameters, then exits program.
 *		-sc: Specifies not to output single cliques (cliques consisting of 1 node)
 *
 * [out]:
 *		All of the maximal cliques written to
 *			cliques.txt
 *
 * [return]:
 * 		Void
 *
 * [notes]:
 * 		vertices = graph nodes
 *
*******************************************************************************/
int main(int argc, const char ** argv) {

	//Begin timing the code
	hipEvent_t start, stop;
	float elapsedTime;
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

	//Default gpu device is 0
	unsigned device = 0;
	//Output cliques that are not connected to the graph? (Cliques of 1)
	bool singleCliques = true;
	//Default input file is adjMatrix.txt
	std::string inFile = "adjMatrix.txt";

	//Set option values for each parameter entered
	for (unsigned i = 0; i < argc; ++i) {
		if (argv[i] == std::string("-v")) {
			//Supply how many vertices are in the input file
			std::stringstream convert(argv[i + 1]);
			convert >> VECTORS;
			i++;

		}else if(argv[i] == std::string("-gpu")){
			//Specify which device to use
			std::stringstream convert(argv[i + 1]);
			convert >> device;
			i++;

		}else if(argv[i] == std::string("-f")){
			//Specify a different input file
			inFile = argv[i + 1];
			i++;

		}else if(argv[i] == std::string("-sc")){
			//Output single cliques?
			singleCliques = false;
			i++;

		}else if(argv[i] == std::string("-help")){
			//Call method to print out all available parameters, then exit program
			printHelp();
			return(0);

		}else if (i > 0) {
			std::cout << "Unknown parameter " << argv[i] << ".\n Use parameter -help for a list of available params." << std::endl;
		}
	}

	//Check for valid vector size
	if (VECTORS < 1) {
		std::cerr
				<< "The number of vectors must be > 0.\n";
		exit(1);
	}

	//Read in the data
	std::fstream inputFile(inFile.c_str(), std::ios_base::in);
	if(inputFile.fail()){
		std::cerr << "Error: Adjacency Matrix file, " << inFile.c_str() << ", could not be found." << std::endl;
		exit(1);
	}


	std::cout << "Reading from " << inFile.c_str() << " ..." << std::endl;

	//Keep track of the highest amount of neighbours any node has
	maxNeighbours = 0;

	//The vertex connected to the most other vertices
	unsigned fixp = 0;
	//Counts how many connections a given vertex has
	unsigned counter = 0;

	//Temp values and counters
	unsigned i = 0, j = 0, x = 0;
	unsigned char c;
	//Read input file to get fixp, and the number of neighbours fixp has
	while(inputFile >> c){
		x = c - 48;
		counter += x;
		//data[j] = data[j] | (x << i % 8);

		i++;
		if(i % 8 == 0 || i % VECTORS == 0){
			if(i % VECTORS == 0){
				if(counter > maxNeighbours){
					maxNeighbours = counter;
					fixp = (i - 1)/ VECTORS;
				}
				counter = 0;
			}
			j++;
		}
	}


	//Get information about the available devices
	unsigned maxTileWidth = 0;
	int nDevices;
		 hipGetDeviceCount(&nDevices);
		  for (int i = 0; i < nDevices; i++) {
		    hipDeviceProp_t prop;
		    hipGetDeviceProperties(&prop, i);
		    //Set maxtilewidth to the max the specified device can handle
		    if(device == i){
		    	maxTileWidth = min(512.0, pow((float)2,floor(log2f(prop.sharedMemPerBlock / ((maxNeighbours) * sizeof(int))))));
		    }
		  }
	std::cout << "Using device " << device << "\n";

	//Declare an integer array for the adjacency values
	//Instead of bit string where each 1 bit represents an adjacency,
	//each vertex has a list of integer ids of other vertices
	//that are adjacent to this one. The first value of each row is
	//a count of the given vertices neighbours
	// -1s represent nulls
	//There is a row for every input vertex, and each row
	// is of length (maxNeighbours + 1) (+1 is for the count)
	int* intData = new int[VECTORS * (maxNeighbours + 1)];
	for(unsigned k = 0; k < VECTORS * (maxNeighbours + 1); ++k){
		intData[k] = -1;
	}

	//Call function to fill in intData with the adjacency values
	generateAdjacencyValues(intData, inFile);



	//Declare structure for holding the initial level of nodes, and allocate to unified memory
	graphNode** structArray = new graphNode*[VECTORS];
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&(structArray), sizeof(structArray) * VECTORS));

	//Call method to generate the first level of nodes (written to structArray)
	// and get the count of those nodes (written to initNodeCount)
	unsigned initNodeCount = generateInitialNodes(structArray, intData, fixp);

	//Call method to perform the maximal clique enumeration
	maximalCliqueEnumeration(structArray, initNodeCount, intData, maxTileWidth, singleCliques);
	std::cout << "Results written to cliques.txt\n";

	//Free memory
	free(intData);

	//Stop recording time, and print the results
	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
	std::cout << "Elapsed time: " << elapsedTime << " ms\n";

	std::cout << "Job's done." << std::endl;
	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line << std::endl;
	exit (1);
}
